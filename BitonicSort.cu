#include "hip/hip_runtime.h"
﻿#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <random>


__global__ void bitonic(int* d_arr, int i, int j) {
	/*int index =  threadIdx.x;*/
	int index = threadIdx.x + (blockDim.x * blockIdx.x);
	int power = i - j + 1;
	int seq_length = pow(2, power);
	int skip = seq_length / 2;
	int x = pow(2, i);

	if (index % seq_length < skip) {
		if ((index / x) % 2 == 0) {
			if (d_arr[index] > d_arr[index + skip]) {
				int temp = d_arr[index];
				d_arr[index] = d_arr[index + skip];
				d_arr[index + skip] = temp;
			}

		}

		else if ((index / x) % 2 == 1) {
			if (d_arr[index] < d_arr[index + skip]) {
				int temp = d_arr[index];
				d_arr[index] = d_arr[index + skip];
				d_arr[index + skip] = temp;
			}
		}
	}
}

double my_log(double x, int base) {
	return log(x) / log(base);
}

void random_ints(int* arr, int N, int count) { // To fill the array
	int i;
	for (i = count; i < N; i++) {
		arr[i] = rand();
	}
}

int main() {
	
	int* arr, *arr_sorted; // Host copies
	int* d_arr; // Device copies
	
	int NumberOfBlocks = 3;
	int NumberOfThreads = 8;
	
	int N = NumberOfBlocks * NumberOfThreads;
	
	// To check if the input is applicable for log 2, if no we will add 0's in the first of array to resolve this issue
	int count = 0;
	while (true) {
		if (my_log(N, 2) > int(my_log(N, 2))) {
			count++;
			N++;
			NumberOfThreads++;
		}
		else {
			break;
		}
	}
	
	int size = N * sizeof(int);

	//Allocate space for device copies
	hipMalloc((void**)&d_arr, size);
	
	//Allocate space for host copies
	arr = (int*)malloc(size);
	arr_sorted = (int*)malloc(size);

	//Fill array
	int x;
	for (x = 0; x < count; x++) {
		arr[x] = 0;
	}
	random_ints(arr, N, count);

	//Copy inputs from host to device
	hipMemcpy(d_arr, arr, size, hipMemcpyHostToDevice);
	
	printf("Original array\n\n");
	int i;
	int k = 0;
	for (i = count; i < N; i++) {
		printf("%d \t", arr[i]);
		if ((k++ + 1) % 5 == 0) {
			printf("\n");
		}
	}

	if (count != 0) {
		printf("\n-----------------------------------\n");
		printf("After we add 0's to make the input applicable for Bitonic sort\n\n");
		for (i = 0; i < N; i++) {
			printf("%d \t", arr[i]);
			if ((i + 1) % 5 == 0) {
				printf("\n");
			}
		}
	}

	//Run bitonic() kernel on GPU
	int j;
	for (i = 1; i <= my_log(N, 2); i++) {
		for (j = 1; j <= i; j++) {
			bitonic << <NumberOfBlocks, NumberOfThreads >> > (d_arr, i, j);
		}
	}

	//Copy result from device to the host
	hipMemcpy(arr_sorted, d_arr, size, hipMemcpyDeviceToHost);
	
	//Print array after the sort is completed
	printf("\n-----------------------------------\n");
	printf("Array after Bitonic Sort\n\n");
	//int i;
	j = 0;
	for (i = count; i < N; i++) {
		printf("%d \t", arr_sorted[i]);
		if ((j++ + 1) % 5 == 0) {
			printf("\n");
		}
	}
	printf("\n-----------------------------------\n");

	free(arr); free(arr_sorted);
	hipFree(d_arr);
	return 0;

}